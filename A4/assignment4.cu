/*
 ============================================================================
 Filename    : assignment4.c
 Author      : Arash Pourhabibi
 ============================================================================
 */

#include <iostream>
#include <iomanip>
#include <fstream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;
#include "utility.h"

void array_process(double *input, double *output, int length, int iterations);
void GPU_array_process(double *input, double *output, int length, int iterations);

int main (int argc, const char *argv[]) {

    int length, iterations;

    if (argc != 3) {
		cout<<"Invalid input!"<<endl<<"Usage: ./assignment4 <length> <iterations>"<<endl;
		return 1;
	} else {
        length      = atoi(argv[1]);
        iterations  = atoi(argv[2]);
        if(length%2!=0)
        {
            cout<<"Invalid input!"<<endl<<"Array length must be even"<<endl;
            return 1;
        }
	}


    //Allocate arrays
    double *input   = new double[length*length];
    double *output  = new double[length*length];

    //Reset Device
    hipDeviceReset();

    //Initialize the arrays
    init(input, length);
    init(output, length);

    //Start timer
    set_clock();

    /*Use either the CPU or the GPU functions*/

    //CPU Baseline
    //Uncomment the block to use the baseline
    array_process(input, output, length, iterations);
    if(iterations%2==0)
    {
        double *temp;
        temp = input;
        input = output;
        output = temp;
    }

    //Stop timer
    double cpu_time = elapsed_time();


    //Save array in file
    save(output, length, "cpu_output.csv");

    //Initialize the arrays
    init(input, length);
    init(output, length);

    double gpu_start = elapsed_time();

    //GPU function
    GPU_array_process(input, output,  length, iterations);

    double gpu_time = elapsed_time();

    //Report time required for n iterations
    cout<<"Running the algorithm on "<<length<<" by "<<length<<" array for "<<iterations<<" iteration"<<endl;
    cout << "takes " << setprecision(4) << cpu_time << "s on the cpu" << endl;
    cout << "takes " << setprecision(4) << gpu_time - gpu_start << "s on the gpu" << endl;
    
    //Save array in file
    save(output, length, "gpu_output.csv");

    //Free allocated memory
    delete[] input;
    delete[] output;

    return 0;
}
