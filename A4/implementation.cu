/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

// copied from CPU baseline
__global__ void GPU_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                printf("(%i, %i) ", i, j);
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;
            }
            printf("\n");
        }
        printf("-----------------------------------------------------\n");
       
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */

    double* gpu_input;
    double* gpu_output;
    // malloc on gpu
    hipMalloc( (void**)&gpu_input, sizeof(double) * length * length);
    hipMalloc( (void**)&gpu_output, sizeof(double) * length* length);

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    // copy data input
    hipMemcpy(
        gpu_input,                  /* DEST */
        input,                      /* SRC */
        sizeof(double) * length * length,    /* NBYTES */
        hipMemcpyHostToDevice      /* DIRECTION */
    );
    
    // copy data output
    // not sure if needed
    hipMemcpy(
        gpu_output,                 /* DEST */
        output,                     /* SRC */
        sizeof(double) * length * length,    /* NBYTES */
        hipMemcpyHostToDevice      /* DIRECTION */
    );

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    dim3 thrsPerBlock(3,4); // 3x4
    dim3 nBlks(2,3); // 2x3
    GPU_process<<<1, 1>>>(gpu_input, gpu_output, length, iterations);

    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    
    // copy output back
    // not sure if needed
    hipMemcpy(
        output,                     /* DEST */
        gpu_output,                 /* SRC */
        sizeof(double) * length * length,    /* NBYTES */
        hipMemcpyDeviceToHost      /* DIRECTION */
    );

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */

    // free gpu_input and gpu_output
    // should cudaFreeArray() be used?
    hipFree(gpu_input);
    hipFree(gpu_output);


    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}