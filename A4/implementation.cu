#include "hip/hip_runtime.h"
/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

// copied from CPU baseline
__global__ void four_ifs(double *input, double *output, int length, int iterations)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y + 1;

    int posInArray = (i)*(length)+(j);

    output[posInArray] = (input[(i-1)*(length)+(j-1)] +
                            input[(i-1)*(length)+(j)]   +
                            input[(i-1)*(length)+(j+1)] +
                            input[(i)*(length)+(j-1)]   +
                            input[(i)*(length)+(j)]     +
                            input[(i)*(length)+(j+1)]   +
                            input[(i+1)*(length)+(j-1)] +
                            input[(i+1)*(length)+(j)]   +
                            input[(i+1)*(length)+(j+1)] ) / 9;
    
    if (posInArray == (length/2-1)*length+(length/2-1))
        output[(length/2-1)*length+(length/2-1)] = 1000;
    
    if (posInArray == (length/2)*length+(length/2-1))
        output[(length/2)*length+(length/2-1)] = 1000;
    
    if (posInArray == (length/2-1)*length+(length/2))
        output[(length/2-1)*length+(length/2)] = 1000;
    
    if (posInArray == (length/2)*length+(length/2))
        output[(length/2)*length+(length/2)] = 1000;
}

__global__ void no_ifs(double *input, double *output, int length, int iterations)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y + 1;

    int posInArray = (i)*(length)+(j);

    output[posInArray] = (input[(i-1)*(length)+(j-1)] +
                            input[(i-1)*(length)+(j)]   +
                            input[(i-1)*(length)+(j+1)] +
                            input[(i)*(length)+(j-1)]   +
                            input[(i)*(length)+(j)]     +
                            input[(i)*(length)+(j+1)]   +
                            input[(i+1)*(length)+(j-1)] +
                            input[(i+1)*(length)+(j)]   +
                            input[(i+1)*(length)+(j+1)] ) / 9;
    
    output[(length/2-1)*length+(length/2-1)] = 1000;
    output[(length/2)*length+(length/2-1)] = 1000;
    output[(length/2-1)*length+(length/2)] = 1000;
    output[(length/2)*length+(length/2)] = 1000;
}

__global__ void one_if(double *input, double *output, int length, int iterations)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y + 1;

    int posInArray = (i)*(length)+(j);

     if (posInArray == (length/2)*length+(length/2) || posInArray == (length/2-1)*length+(length/2-1) || posInArray == (length/2)*length+(length/2-1) || posInArray == (length/2-1)*length+(length/2)){
        output[posInArray] = 1000;
        return;
     }

    output[posInArray] = (input[(i-1)*(length)+(j-1)] +
                            input[(i-1)*(length)+(j)]   +
                            input[(i-1)*(length)+(j+1)] +
                            input[(i)*(length)+(j-1)]   +
                            input[(i)*(length)+(j)]     +
                            input[(i)*(length)+(j+1)]   +
                            input[(i+1)*(length)+(j-1)] +
                            input[(i+1)*(length)+(j)]   +
                            input[(i+1)*(length)+(j+1)] ) / 9;
    
}
__global__ void no_rewrite(double *input, double *output, int length, int iterations)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y + 1;

    int posInArray = (i)*(length)+(j);

     if (posInArray == (length/2)*length+(length/2) || posInArray == (length/2-1)*length+(length/2-1) || posInArray == (length/2)*length+(length/2-1) || posInArray == (length/2-1)*length+(length/2)){
        // output[posInArray] = 1000;
        return;
     }

    output[posInArray] = (input[(i-1)*(length)+(j-1)] +
                            input[(i-1)*(length)+(j)]   +
                            input[(i-1)*(length)+(j+1)] +
                            input[(i)*(length)+(j-1)]   +
                            input[(i)*(length)+(j)]     +
                            input[(i)*(length)+(j+1)]   +
                            input[(i+1)*(length)+(j-1)] +
                            input[(i+1)*(length)+(j)]   +
                            input[(i+1)*(length)+(j+1)] ) / 9;
    
}


// not working for more than 76 length
__global__ void shared_memory(double *input, double *output, int length, int iterations)
{
    int max_len = length * length;

    int tidx = threadIdx.x + 1;
    int tidy = threadIdx.y + 1;

    int arrX = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
    int arrY = (blockIdx.y * blockDim.y) + threadIdx.y + 1;

    int posInArray = arrX + length * arrY;

    // in shared memory
    __shared__ double shared_input[THREADS_PER_BLOCK + 2][THREADS_PER_BLOCK + 2];
    // extern __shared__ double shared_output[64][n];
    // copy data
    
    // shared_input[y][x]
    shared_input[tidy][tidx] = (posInArray < max_len) ? input[posInArray] : 0;
    if (arrX >= length - 1 || arrY >= length - 1) return;
    // printf("%d, %d\n", arrX, arrY);

    if (tidx == 1){
        shared_input[tidy][0] = input[posInArray - 1];
        if(tidy == 1){
            shared_input[0][0] = input[posInArray - 1 - length];
        }
    }
    if(tidy == 1){
        shared_input[0][tidx] = input[posInArray - length];
        if (tidx == THREADS_PER_BLOCK){
            shared_input[0][THREADS_PER_BLOCK + 1] = input[posInArray + 1 - length];
        }
    }
    if(tidx == THREADS_PER_BLOCK){
        shared_input[tidy][THREADS_PER_BLOCK + 1] = (posInArray + 1 < max_len) ? input[posInArray + 1] : 0;
        if(tidy == THREADS_PER_BLOCK){
            shared_input[THREADS_PER_BLOCK + 1][THREADS_PER_BLOCK + 1] = (posInArray + 1 + length < max_len) ? input[posInArray + 1 + length] : 0;
        }
    }
    if (tidy == THREADS_PER_BLOCK){
            shared_input[THREADS_PER_BLOCK + 1][tidx] = (posInArray + length < max_len) ? input[posInArray + length] : 0;
            if(tidx == 1){
                shared_input[THREADS_PER_BLOCK + 1][0] = (posInArray - 1 + length < max_len) ? input[posInArray - 1 + length] : 0;
            }
    }
    __syncthreads();

    if (posInArray == (length/2-1)*length+(length/2-1) ||
        posInArray == (length/2)*length+(length/2-1)   ||
        posInArray == (length/2-1)*length+(length/2)   ||
        posInArray == (length/2)*length+(length/2))
    {
        output[posInArray] = 1000;
        // __syncthreads();
        return;
    }

    output[posInArray] = (
                            shared_input[tidy-1][tidx-1]    +
                            shared_input[tidy-1][tidx  ]    +
                            shared_input[tidy-1][tidx+1]    +
                            shared_input[tidy  ][tidx-1]    +
                            shared_input[tidy  ][tidx  ]    +
                            shared_input[tidy  ][tidx+1]    +
                            shared_input[tidy+1][tidx-1]    +
                            shared_input[tidy+1][tidx  ]    +
                            shared_input[tidy+1][tidx+1]
                        ) / 9;
    __syncthreads();

    // if(tidx == 1 && tidy == 1 && blockIdx.x == 0 && blockIdx.y == 0){
    //     printf("shared -------------------------------------\n");
    //     for(int y = 0; y < THREADS_PER_BLOCK + 2; y++){
    //         for(int x = 0; x < THREADS_PER_BLOCK + 2; x++){
    //             printf("%lf, ", shared_input[y][x]);
    //         }
    //         printf("\n");
    //     }

    //     printf("output -------------------------------------\n");
    //     for(int y = 0; y < length; y++){
    //         for(int x = 0; x < length; x++){
    //             printf("%lf, ", output[x + length * y]);
    //         }
    //         printf("\n");
    //     }
    // }
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */

    double* gpu_input;
    double* gpu_output;
    // malloc on gpu
    hipMalloc( (void**)&gpu_input, sizeof(double) * length * length);
    hipMalloc( (void**)&gpu_output, sizeof(double) * length* length);

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    // copy data input
    hipMemcpy(
        gpu_input,                  /* DEST */
        input,                      /* SRC */
        sizeof(double) * length * length,    /* NBYTES */
        hipMemcpyHostToDevice      /* DIRECTION */
    );
    
    // copy data output
    // not sure if needed
    hipMemcpy(
        gpu_output,                 /* DEST */
        output,                     /* SRC */
        sizeof(double) * length * length,    /* NBYTES */
        hipMemcpyHostToDevice      /* DIRECTION */
    );

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    hipEventRecord(comp_start);

    /* GPU calculation goes here */
    int thrsPerBlock = 8; 
    int nBlks = ceil((double)(length -2)/thrsPerBlock);

    dim3 thrsPerBlockDim(thrsPerBlock, thrsPerBlock);
    dim3 nBlksDim(nBlks, nBlks);

    for(int n=0; n<iterations; n++)
    {
        one_if<<<nBlksDim, thrsPerBlockDim>>>(gpu_input, gpu_output, length, iterations);

        hipDeviceSynchronize();
        
        double *temp;
        temp = gpu_input;
        gpu_input = gpu_output;
        gpu_output = temp;
    }
    
    // swap back
    double *temp;
    temp = gpu_input;
    gpu_input = gpu_output;
    gpu_output = temp;

    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    
    // copy output back
    // not sure if needed
    hipMemcpy(
        output,                     /* DEST */
        gpu_output,                 /* SRC */
        sizeof(double) * length * length,    /* NBYTES */
        hipMemcpyDeviceToHost      /* DIRECTION */
    );

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */

    // free gpu_input and gpu_output
    // should hipFreeArray() be used?
    hipFree(gpu_input);
    hipFree(gpu_output);


    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}