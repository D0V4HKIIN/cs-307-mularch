/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

// copied from CPU baseline
__global__ void four_ifs(double *input, double *output, int length, int iterations)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y + 1;

    int posInArray = (i)*(length)+(j);

    output[posInArray] = (input[(i-1)*(length)+(j-1)] +
                            input[(i-1)*(length)+(j)]   +
                            input[(i-1)*(length)+(j+1)] +
                            input[(i)*(length)+(j-1)]   +
                            input[(i)*(length)+(j)]     +
                            input[(i)*(length)+(j+1)]   +
                            input[(i+1)*(length)+(j-1)] +
                            input[(i+1)*(length)+(j)]   +
                            input[(i+1)*(length)+(j+1)] ) / 9;
    
    if (posInArray == (length/2-1)*length+(length/2-1))
        output[(length/2-1)*length+(length/2-1)] = 1000;
    
    if (posInArray == (length/2)*length+(length/2-1))
        output[(length/2)*length+(length/2-1)] = 1000;
    
    if (posInArray == (length/2-1)*length+(length/2))
        output[(length/2-1)*length+(length/2)] = 1000;
    
    if (posInArray == (length/2)*length+(length/2))
        output[(length/2)*length+(length/2)] = 1000;
}

__global__ void no_ifs(double *input, double *output, int length, int iterations)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y + 1;

    int posInArray = (i)*(length)+(j);

    output[posInArray] = (input[(i-1)*(length)+(j-1)] +
                            input[(i-1)*(length)+(j)]   +
                            input[(i-1)*(length)+(j+1)] +
                            input[(i)*(length)+(j-1)]   +
                            input[(i)*(length)+(j)]     +
                            input[(i)*(length)+(j+1)]   +
                            input[(i+1)*(length)+(j-1)] +
                            input[(i+1)*(length)+(j)]   +
                            input[(i+1)*(length)+(j+1)] ) / 9;
    
    output[(length/2-1)*length+(length/2-1)] = 1000;
    output[(length/2)*length+(length/2-1)] = 1000;
    output[(length/2-1)*length+(length/2)] = 1000;
    output[(length/2)*length+(length/2)] = 1000;
}

__global__ void GPU_process(double *input, double *output, int length, int iterations)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y + 1;

    int posInArray = (i)*(length)+(j);

     if (posInArray == (length/2)*length+(length/2) || posInArray == (length/2-1)*length+(length/2-1) || posInArray == (length/2)*length+(length/2-1) || posInArray == (length/2-1)*length+(length/2)){
        output[posInArray] = 1000;
        return;
     }

    output[posInArray] = (input[(i-1)*(length)+(j-1)] +
                            input[(i-1)*(length)+(j)]   +
                            input[(i-1)*(length)+(j+1)] +
                            input[(i)*(length)+(j-1)]   +
                            input[(i)*(length)+(j)]     +
                            input[(i)*(length)+(j+1)]   +
                            input[(i+1)*(length)+(j-1)] +
                            input[(i+1)*(length)+(j)]   +
                            input[(i+1)*(length)+(j+1)] ) / 9;
    
   
    
    
}





// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */

    double* gpu_input;
    double* gpu_output;
    // malloc on gpu
    hipMalloc( (void**)&gpu_input, sizeof(double) * length * length);
    hipMalloc( (void**)&gpu_output, sizeof(double) * length* length);

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    // copy data input
    hipMemcpy(
        gpu_input,                  /* DEST */
        input,                      /* SRC */
        sizeof(double) * length * length,    /* NBYTES */
        hipMemcpyHostToDevice      /* DIRECTION */
    );
    
    // copy data output
    // not sure if needed
    hipMemcpy(
        gpu_output,                 /* DEST */
        output,                     /* SRC */
        sizeof(double) * length * length,    /* NBYTES */
        hipMemcpyHostToDevice      /* DIRECTION */
    );

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    hipEventRecord(comp_start);

    /* GPU calculation goes here */
    int thrsPerBlock = 8; 
    int nBlks = ceil((double)(length -2)/thrsPerBlock);

    dim3 thrsPerBlockDim(thrsPerBlock, thrsPerBlock);
    dim3 nBlksDim(nBlks, nBlks);

    for(int n=0; n<iterations; n++)
    {
        GPU_process<<<nBlksDim, thrsPerBlockDim>>>(gpu_input, gpu_output, length, iterations);

        hipDeviceSynchronize();
        
        double *temp;
        temp = gpu_input;
        gpu_input = gpu_output;
        gpu_output = temp;
    }
    
    // swap back
    double *temp;
    temp = gpu_input;
    gpu_input = gpu_output;
    gpu_output = temp;

    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    
    // copy output back
    // not sure if needed
    hipMemcpy(
        output,                     /* DEST */
        gpu_output,                 /* SRC */
        sizeof(double) * length * length,    /* NBYTES */
        hipMemcpyDeviceToHost      /* DIRECTION */
    );

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */

    // free gpu_input and gpu_output
    // should cudaFreeArray() be used?
    hipFree(gpu_input);
    hipFree(gpu_output);


    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}